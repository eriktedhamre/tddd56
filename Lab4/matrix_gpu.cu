#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>
#include "milli.h"

__global__
void add_matrix(float *a, float *b, float *c, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	//printf("%d \n",idx * N + idy);
	c[idx * N + idy] = a[idx * N + idy] + b[idx * N + idy];
	//c[idy * N + idx] = a[idy * N + idx] + b[idy * N + idx];
/*
// Bad pattern to access after each other since each thread has to read memory for each access and threads can not use one read for multiple data points
	for (int i = 0; i < N; i++) {
			c[idx * N + i] = a[idx * N + i] + b[idx * N + i];
			//c[idx + N * i] = a[idx + N * i] + b[idx + N * i];
	}
*/
}
/*
hipMalloc( (void**)&cd, size );
dim3 dimBlock( blocksize, 1 );
dim3 dimGrid( 1, 1 );
simple<<<dimGrid, dimBlock>>>(cd);
hipDeviceSynchronize();
hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );
hipFree( cd );
*/

void add_matrix_cpu(float *a, float *b, float *c, int N)
{
	int index;

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			index = i + j*N;
			c[index] = a[index] + b[index];
		}
}

int main()
{
	const int N = 6144 ;
	const int BLOCKSIZE = 32;
	const int GRIDSIZE = 192;
	long size = (long) N*N*sizeof(float);
	float * a_h,* a_d,* b_h,* b_d,* c_d,* c_h;
	a_h = (float *)malloc(size);
	hipMalloc((void **) &a_d, size);
	b_h = (float *)malloc(size);
	hipMalloc((void **) &b_d, size);
	c_h = (float *)malloc(size);
	hipMalloc((void **) &c_d, size);

	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
		{
			a_h[i+j*N] = 10 + i;
			b_h[i+j*N] = (float)j / N;
			c_h[i+j*N] = 0;
		}
	}
	float GPUTime;
		hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
		hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
		hipMemcpy(c_d, c_h, size, hipMemcpyHostToDevice);

  hipEvent_t start;
		hipEvent_t stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		dim3 dimBlock( BLOCKSIZE, BLOCKSIZE );
		dim3 dimGrid( GRIDSIZE, GRIDSIZE );
		hipEventRecord(start, 0);
		add_matrix<<<dimGrid, dimBlock>>>(a_d, b_d, c_d, N);
		hipEventRecord(stop, 0);
		hipDeviceSynchronize();
		hipEventSynchronize(start);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&GPUTime, start, stop);

		hipError_t err = hipGetLastError();
	  	if (err != hipSuccess)
	    	printf("Error: %s\n", hipGetErrorString(err));
		hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

		float * a, *b, *c;
		a = (float *)malloc(size);
		b = (float *)malloc(size);
		c = (float *)malloc(size);

		for (int i = 0; i < N; i++){
			for (int j = 0; j < N; j++)
			{
				a[i+j*N] = 10 + i;
				b[i+j*N] = (float)j /N;
				c[i+j*N] = 0;
			}
		}
		int CPUTime;
		GetMilliseconds();
		add_matrix_cpu(a, b, c, N);
		CPUTime = GetMilliseconds();



	for (int i = 0; i < N; i+=1000)
	{
		for (int j = 0; j < N; j+=1000)
		{
			printf("%0.2f ", c_h[i+j*N]);
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}

	printf("GPU time = %f ms\n", GPUTime);
	printf("CPU time = %d ms\n", CPUTime);

}
