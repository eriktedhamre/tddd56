#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10


__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
  // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

  int dy, dx;
  unsigned int sumx, sumy, sumz;

  int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!

	if (x < imagesizex && y < imagesizey) // If inside image
	{
// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)
		{
			// Use max and min to avoid branching!
			int yy = min(max(y+dy, 0), imagesizey-1);
			int xx = min(max(x+dx, 0), imagesizex-1);

			sumx += image[((yy)*imagesizex+(xx))*3+0];
			sumy += image[((yy)*imagesizex+(xx))*3+1];
			sumz += image[((yy)*imagesizex+(xx))*3+2];
		}
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
	}
}

__global__ void filterOptimized(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
  // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

  int dy, dx;
  unsigned int sumx, sumy, sumz;

  int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!
  extern __shared__ unsigned char sData[]; //size = (2*kernelsizex+1)*(2*kernelsizey+1)*3*sizeof(unsigned char)
// use threadIdx.x and .y insetad of x and y since we are in a block that is executed on a MP?
/*
  sData[(y*imagesizex+x)*3+0] = image[(y*imagesizex+x)*3+0];
  sData[(y*imagesizex+x)*3+1] = image[(y*imagesizex+x)*3+1];
  sData[(y*imagesizex+x)*3+2] = image[(y*imagesizex+x)*3+2];

  sData[((2*kernelsizex+1)*kernelsizey+1)*(y*imagesizex+x)*3+0] = image[(y*imagesizex+x)*3+0];
  sData[((2*kernelsizex+1)*kernelsizey+1)*(y*imagesizex+x)*3+1] = image[(y*imagesizex+x)*3+1];
  sData[((2*kernelsizex+1)*kernelsizey+1)*(y*imagesizex+x)*3+2] = image[(y*imagesizex+x)*3+2];
*/

// want to get x-threadidx and y-threadidy to get the values outside of the used grid
  int imy = min(max(y-2*threadIdx.y,0), imagesizey-1);
  int imx = min(max(x-2*threadIdx.x,0), imagesizex-1);
  sData[(threadIdx.y*kernelsizex+threadIdx.x)*3+0] = image[(imy*imagesizex+x-2*imx)*3+0];
  sData[(threadIdx.y*kernelsizex+threadIdx.x)*3+1] = image[(imy*imagesizex+x-2*imx)*3+1];
  sData[(threadIdx.y*kernelsizex+threadIdx.x)*3+2] = image[(imy*imagesizex+x-2*imx)*3+2];

  imy = min(max(y*imagesizex, 0), imagesizey-1);
  imx = min(max(x,0), imagesizex-1);
  sData[((2*kernelsizex+1)*kernelsizey+1)+(threadIdx.y*kernelsizex+threadIdx.x)*3+0] = image[(imy*imagesizex+imx)*3+0];
  sData[((2*kernelsizex+1)*kernelsizey+1)+(threadIdx.y*kernelsizex+threadIdx.x)*3+1] = image[(imy*imagesizex+imx)*3+1];
  sData[((2*kernelsizex+1)*kernelsizey+1)+(threadIdx.y*kernelsizex+threadIdx.x)*3+2] = image[(imy*imagesizex+imx)*3+2];

  __syncthreads();
  if (x < imagesizex && y < imagesizey) // If inside image
	{
// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=0;dy<=2*kernelsizey+1;dy++)
		for(dx=0;dx<=2*kernelsizex+1;dx++)
		{
			// Use max and min to avoid branching!
			int yy = min(max(threadIdx.y+dy, 0), 2*kernelsizey);
			int xx = min(max(threadIdx.x+dx, 0), 2*kernelsizex);

			sumx += sData[((yy)*imagesizex+(xx))*3+0];
			sumy += sData[((yy)*imagesizex+(xx))*3+1];
			sumz += sData[((yy)*imagesizex+(xx))*3+2];
		}
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
    __syncthreads();
	}
}


// Global variables for image data

unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}

	pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);
	dim3 grid(imagesizex,imagesizey);
  filter<<<grid,1>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // Awful load balance
	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
  unsigned char* sharedData;
  hipMalloc((void**)&sharedData, (kernelsizex*2+1)*(kernelsizey*2+1)*3*sizeof(unsigned char));
  filterOptimized<<<grid, 1, (kernelsizex*2+1)*(kernelsizey*2+1)*3*sizeof(unsigned char)>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey);
  hipDeviceSynchronize();
  //	Check for errors!
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Optimized version Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
  hipFree( sharedData );
  hipFree( dev_bitmap );
	hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(2, 2);

// You can save the result to a file like this:
//	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
