#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10


__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
  // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

  int dy, dx;
  unsigned int sumx, sumy, sumz;

  int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!

	if (x < imagesizex && y < imagesizey) // If inside image
	{
// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)
		{
			// Use max and min to avoid branching!
			int yy = min(max(y+dy, 0), imagesizey-1);
			int xx = min(max(x+dx, 0), imagesizex-1);

			sumx += image[((yy)*imagesizex+(xx))*3+0];
			sumy += image[((yy)*imagesizex+(xx))*3+1];
			sumz += image[((yy)*imagesizex+(xx))*3+2];
		}
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
	}
}

__global__ void filterOptimized(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
  // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	//printf("blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d, blockIdx.y = %d, blockDim.y = %d, threadIdx.y = %d\n", blockIdx.x, blockDim.x, threadIdx.x, blockIdx.y, blockDim.y, threadIdx.y);

  int dy, dx;
  unsigned int sumx, sumy, sumz;

  //int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!
		int divby = (2*kernelsizex+1)*(2*kernelsizey+1);
  //extern __shared__ unsigned char sData[]; //size = (2*kernelsizex+1)*(2*kernelsizey+1)*3*sizeof(unsigned char)
		__shared__ unsigned char sData[(2*maxKernelSizeX+1)*(2*maxKernelSizeY+1)*sizeof(unsigned char)*3];
/*
for (size_t b = 0; b < blockDim; b++) {
		from image to local image
		// x koordinat för lokal tråd, ii = thread.x*blockDim
		// y koordinat för lokal tråd jj =


}
*/
// want to get x-threadIdx and y-threadIdy to get the values outside of the used grid

		for (int i = -maxKernelSizeY; i <= maxKernelSizeY; i+=1) {
			int ii = min(max(y+i,0),imagesizey-1);
			for (int j = -maxKernelSizeX; j <= maxKernelSizeX; j+=1) {
				int jj = min(max(x+j,0), imagesizex-1);
				if(x==1 && y==1){
					printf("x = %d, y = %d, ii = %d, i=%d, jj = %d, j = %d \n", x, y, ii, i, jj, j);
				}
				sData[((i+maxKernelSizeY)*(2*maxKernelSizeX+1)+j+maxKernelSizeX)*3+0] = image[(ii * imagesizex + jj)*3+0];
				sData[((i+maxKernelSizeY)*(2*maxKernelSizeX+1)+j+maxKernelSizeX)*3+1] = image[(ii * imagesizex + jj)*3+1];
				sData[((i+maxKernelSizeY)*(2*maxKernelSizeX+1)+j+maxKernelSizeX)*3+2] = image[(ii * imagesizex + jj)*3+2];
			}
		}

  __syncthreads();
		/*
		if(x == 1 && y == 1){
			for (int i = 0; i < 2*maxKernelSizeY+1; i++) {
				for (int j = 0; j < (2*maxKernelSizeX+1)*3; j++) {
					printf("%d ",sData[i * (2*maxKernelSizeY+1) + j]);
				}
				printf("\n");
			}
		}
		*/
  if (x < imagesizex && y < imagesizey) // If inside image
	{
// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)
		{
			// Use max and min to avoid branching!
			int yy = min(max(threadIdx.y+dy, 0), 2*maxKernelSizeY);
			int xx = min(max(threadIdx.x+dx, 0), 2*maxKernelSizeX);

			sumx += sData[((yy)*maxKernelSizeX+(xx))*3+0];
			sumy += sData[((yy)*maxKernelSizeX+(xx))*3+1];
			sumz += sData[((yy)*maxKernelSizeX+(xx))*3+2];
		}
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
    __syncthreads();
	}
}


// Global variables for image data

unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}


	float GPUTime, OptimizedGPUTime;
	pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);
	dim3 grid(imagesizex,imagesizey);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
 filter<<<grid,1>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // Awful load balance
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&GPUTime, start, stop);
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
		//hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
		printf("calling filterOptimized \n");
		hipEventRecord(start, 0);
		dim3 block(2,2);
  filterOptimized<<<grid, block>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey);
		hipEventRecord(stop, 0);
		hipDeviceSynchronize();
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );

		hipEventElapsedTime(&OptimizedGPUTime, start, stop);
		printf("GPU time = %f ms, Optimized GPU time = %f ms\n", GPUTime, OptimizedGPUTime);
  //	Check for errors!
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("Optimized version Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();
	hipFree(dev_input);
	hipFree(dev_bitmap);

}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();
/*
	computeImages(1, 1);
	computeImages(2, 2);
	computeImages(3, 3);
	computeImages(4, 4);
	computeImages(5, 5);
	computeImages(6, 6);
	computeImages(7, 7);
	computeImages(8, 8);
	computeImages(9, 9);
	*/
	computeImages(2, 2);


// You can save the result to a file like this:
//	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
